#include "hip/hip_runtime.h"
// Copyright 2014-2017 Oxford University Innovation Limited and the authors of InfiniTAM

#include "ITMLowLevelEngine_CUDA.h"

#include "../Shared/ITMLowLevelEngine_Shared.h"
#include "../../../Utils/ITMCUDAUtils.h"
#include <ORUtils/CUDADefines.h>

using namespace ITMLib;

ITMLowLevelEngine_CUDA::ITMLowLevelEngine_CUDA(void)
{
	ORcudaSafeCall(hipMalloc((void**)&counterTempData_device, sizeof(int)));
	ORcudaSafeCall(hipHostMalloc((void**)&counterTempData_host, sizeof(int)));
}

ITMLowLevelEngine_CUDA::~ITMLowLevelEngine_CUDA(void)
{
	ORcudaSafeCall(hipFree(counterTempData_device));
	ORcudaSafeCall(hipHostFree(counterTempData_host));
}

__global__ void convertColourToIntensity_device(float *imageData_out, Vector2i dims, const Vector4u *imageData_in);

__global__ void boxFilter2x2_device(float *imageData_out, const float *imageData_in, Vector2i dims);

__global__ void filterSubsample_device(float *imageData_out, Vector2i newDims, const float *imageData_in, Vector2i oldDims);
__global__ void filterSubsample_device(Vector4u *imageData_out, Vector2i newDims, const Vector4u *imageData_in, Vector2i oldDims);

__global__ void filterSubsampleWithHoles_device(float *imageData_out, Vector2i newDims, const float *imageData_in, Vector2i oldDims);
__global__ void filterSubsampleWithHoles_device(Vector4f *imageData_out, Vector2i newDims, const Vector4f *imageData_in, Vector2i oldDims);

__global__ void kernelBilateral5x5( Vector2i imgSize, const float *data, float* filtered_data, float depthUncertaintyCoef);

__global__ void gradientX_device(Vector4s *grad, const Vector4u *image, Vector2i imgSize);
__global__ void gradientY_device(Vector4s *grad, const Vector4u *image, Vector2i imgSize);
__global__ void gradientXY_device(Vector2f *grad, const float *image, Vector2i imgSize);

__global__ void countValidDepths_device(const float *imageData_in, int imgSizeTotal, int *counterTempData_device);

// host methods

void ITMLowLevelEngine_CUDA::CopyImage(ITMUChar4Image *image_out, const ITMUChar4Image *image_in) const
{
	Vector4u *dest = image_out->GetData(MEMORYDEVICE_CUDA);
	const Vector4u *src = image_in->GetData(MEMORYDEVICE_CUDA);

	ORcudaSafeCall(hipMemcpy(dest, src, image_in->dataSize * sizeof(Vector4u), hipMemcpyDeviceToDevice));
}

void ITMLowLevelEngine_CUDA::CopyImage(ITMFloatImage *image_out, const ITMFloatImage *image_in) const
{
	float *dest = image_out->GetData(MEMORYDEVICE_CUDA);
	const float *src = image_in->GetData(MEMORYDEVICE_CUDA);

	ORcudaSafeCall(hipMemcpy(dest, src, image_in->dataSize * sizeof(float), hipMemcpyDeviceToDevice));
}

void ITMLowLevelEngine_CUDA::CopyImage(ITMFloat4Image *image_out, const ITMFloat4Image *image_in) const
{
	Vector4f *dest = image_out->GetData(MEMORYDEVICE_CUDA);
	const Vector4f *src = image_in->GetData(MEMORYDEVICE_CUDA);

	ORcudaSafeCall(hipMemcpy(dest, src, image_in->dataSize * sizeof(Vector4f), hipMemcpyDeviceToDevice));
}

void ITMLowLevelEngine_CUDA::ConvertColourToIntensity(ITMFloatImage *image_out, const ITMUChar4Image *image_in) const
{
	const Vector2i dims = image_in->noDims;
	image_out->ChangeDims(dims);

	float *dest = image_out->GetData(MEMORYDEVICE_CUDA);
	const Vector4u *src = image_in->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)dims.x / (float)blockSize.x), (int)ceil((float)dims.y / (float)blockSize.y));

	convertColourToIntensity_device << <gridSize, blockSize >> >(dest, dims, src);
	ORcudaKernelCheck;
}

void ITMLowLevelEngine_CUDA::FilterIntensity(ITMFloatImage *image_out, const ITMFloatImage *image_in) const
{
	Vector2i dims = image_in->noDims;

	image_out->ChangeDims(dims);
	image_out->Clear(0);

	const float *imageData_in = image_in->GetData(MEMORYDEVICE_CUDA);
	float *imageData_out = image_out->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)dims.x / (float)blockSize.x), (int)ceil((float)dims.y / (float)blockSize.y));

	boxFilter2x2_device << <gridSize, blockSize >> >(imageData_out, imageData_in, dims);
	ORcudaKernelCheck;
}

void ITMLowLevelEngine_CUDA::FilterSubsample(ITMUChar4Image *image_out, const ITMUChar4Image *image_in) const
{
	Vector2i oldDims = image_in->noDims;
	Vector2i newDims; newDims.x = image_in->noDims.x / 2; newDims.y = image_in->noDims.y / 2;

	image_out->ChangeDims(newDims);

	const Vector4u *imageData_in = image_in->GetData(MEMORYDEVICE_CUDA);
	Vector4u *imageData_out = image_out->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)newDims.x / (float)blockSize.x), (int)ceil((float)newDims.y / (float)blockSize.y));

	filterSubsample_device << <gridSize, blockSize >> >(imageData_out, newDims, imageData_in, oldDims);
	ORcudaKernelCheck;
}

void ITMLowLevelEngine_CUDA::FilterSubsample(ITMFloatImage *image_out, const ITMFloatImage *image_in) const
{
	Vector2i oldDims = image_in->noDims;
	Vector2i newDims; newDims.x = image_in->noDims.x / 2; newDims.y = image_in->noDims.y / 2;

	image_out->ChangeDims(newDims);
	image_out->Clear(0);

	const float *imageData_in = image_in->GetData(MEMORYDEVICE_CUDA);
	float *imageData_out = image_out->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)newDims.x / (float)blockSize.x), (int)ceil((float)newDims.y / (float)blockSize.y));

	filterSubsample_device << <gridSize, blockSize >> >(imageData_out, newDims, imageData_in, oldDims);
	ORcudaKernelCheck;
}

void ITMLowLevelEngine_CUDA::FilterSubsampleWithHoles(ITMFloatImage *image_out, const ITMFloatImage *image_in) const
{
	Vector2i oldDims = image_in->noDims;
	Vector2i newDims; newDims.x = image_in->noDims.x / 2; newDims.y = image_in->noDims.y / 2;

	image_out->ChangeDims(newDims);

	const float *imageData_in = image_in->GetData(MEMORYDEVICE_CUDA);
	float *imageData_out = image_out->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)newDims.x / (float)blockSize.x), (int)ceil((float)newDims.y / (float)blockSize.y));

	filterSubsampleWithHoles_device << <gridSize, blockSize >> >(imageData_out, newDims, imageData_in, oldDims);
	ORcudaKernelCheck;
}

void ITMLowLevelEngine_CUDA::FilterSubsampleWithHoles(ITMFloat4Image *image_out, const ITMFloat4Image *image_in) const
{
	Vector2i oldDims = image_in->noDims;
	Vector2i newDims; newDims.x = image_in->noDims.x / 2; newDims.y = image_in->noDims.y / 2;

	image_out->ChangeDims(newDims);

	const Vector4f *imageData_in = image_in->GetData(MEMORYDEVICE_CUDA);
	Vector4f *imageData_out = image_out->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)newDims.x / (float)blockSize.x), (int)ceil((float)newDims.y / (float)blockSize.y));

	filterSubsampleWithHoles_device << <gridSize, blockSize >> >(imageData_out, newDims, imageData_in, oldDims);
	ORcudaKernelCheck;
}

void ITMLowLevelEngine_CUDA::BilateralFilter5x5(ITMFloatImage *img, float depthUncertaintyCoef) const
{
    img->UpdateDeviceFromHost();
    ITMFloatImage tmp(*img);
    dim3 threads(16, 16);
    dim3 blocks( (img->noDims.x + threads.x-1) / threads.x, (img->noDims.y + threads.y-1) / threads.y );
    kernelBilateral5x5<<<blocks, threads>>>( img->noDims, tmp.GetDataConst(MEMORYDEVICE_CUDA), img->GetData(MEMORYDEVICE_CUDA), depthUncertaintyCoef);
    hipDeviceSynchronize();
    img->UpdateHostFromDevice();
}

void ITMLowLevelEngine_CUDA::GradientX(ITMShort4Image *grad_out, const ITMUChar4Image *image_in) const
{
	grad_out->ChangeDims(image_in->noDims);
	Vector2i imgSize = image_in->noDims;

	Vector4s *grad = grad_out->GetData(MEMORYDEVICE_CUDA);
	const Vector4u *image = image_in->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)imgSize.x / (float)blockSize.x), (int)ceil((float)imgSize.y / (float)blockSize.y));

	ORcudaSafeCall(hipMemset(grad, 0, imgSize.x * imgSize.y * sizeof(Vector4s)));

	gradientX_device << <gridSize, blockSize >> >(grad, image, imgSize);
	ORcudaKernelCheck;
}

void ITMLowLevelEngine_CUDA::GradientY(ITMShort4Image *grad_out, const ITMUChar4Image *image_in) const
{
	grad_out->ChangeDims(image_in->noDims);
	Vector2i imgSize = image_in->noDims;

	Vector4s *grad = grad_out->GetData(MEMORYDEVICE_CUDA);
	const Vector4u *image = image_in->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)imgSize.x / (float)blockSize.x), (int)ceil((float)imgSize.y / (float)blockSize.y));

	ORcudaSafeCall(hipMemset(grad, 0, imgSize.x * imgSize.y * sizeof(Vector4s)));

	gradientY_device << <gridSize, blockSize >> >(grad, image, imgSize);
	ORcudaKernelCheck;
}

void ITMLowLevelEngine_CUDA::GradientXY(ITMFloat2Image *grad_out, const ITMFloatImage *image_in) const
{
	Vector2i imgSize = image_in->noDims;
	grad_out->ChangeDims(imgSize);
	grad_out->Clear();

	Vector2f *grad = grad_out->GetData(MEMORYDEVICE_CUDA);
	const float *image = image_in->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)imgSize.x / (float)blockSize.x), (int)ceil((float)imgSize.y / (float)blockSize.y));

	gradientXY_device << <gridSize, blockSize >> >(grad, image, imgSize);
	ORcudaKernelCheck;
}

int ITMLowLevelEngine_CUDA::CountValidDepths(const ITMFloatImage *image_in) const
{
	const float *imageData_in = image_in->GetData(MEMORYDEVICE_CUDA);
	Vector2i imgSize = image_in->noDims;

	dim3 blockSize(256);
	dim3 gridSize((int)ceil((float)imgSize.x*imgSize.y / (float)blockSize.x));

	ORcudaSafeCall(hipMemset(counterTempData_device, 0, sizeof(int)));
	countValidDepths_device <<<gridSize, blockSize>>>(imageData_in, imgSize.x*imgSize.y, counterTempData_device);
	ORcudaKernelCheck;
	ORcudaSafeCall(hipMemcpy(counterTempData_host, counterTempData_device, sizeof(int), hipMemcpyDeviceToHost));

	return *counterTempData_host;
}

// device functions

__global__ void convertColourToIntensity_device(float *imageData_out, Vector2i dims, const Vector4u *imageData_in)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x > dims.x - 1 || y > dims.y - 1) return;

	convertColourToIntensity(imageData_out, x, y, dims, imageData_in);
}

__global__ void boxFilter2x2_device(float *imageData_out, const float *imageData_in, Vector2i dims)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= dims.x - 2 || y >= dims.y - 2 || x <= 1 || y <= 1) return;

	boxFilter2x2(imageData_out, x, y, dims, imageData_in, x, y, dims);
}

__global__ void filterSubsample_device(Vector4u *imageData_out, Vector2i newDims, const Vector4u *imageData_in, Vector2i oldDims)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x > newDims.x - 1 || y > newDims.y - 1) return;

	filterSubsample(imageData_out, x, y, newDims, imageData_in, oldDims);
}

__global__ void filterSubsample_device(float *imageData_out, Vector2i newDims, const float *imageData_in, Vector2i oldDims)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x > newDims.x - 2 || y > newDims.y - 2 || x < 1 || y < 1) return;

	boxFilter2x2(imageData_out, x, y, newDims, imageData_in, x * 2, y * 2, oldDims);
}

__global__ void filterSubsampleWithHoles_device(float *imageData_out, Vector2i newDims, const float *imageData_in, Vector2i oldDims)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x > newDims.x - 1 || y > newDims.y - 1) return;

	filterSubsampleWithHoles(imageData_out, x, y, newDims, imageData_in, oldDims);
}

__global__ void filterSubsampleWithHoles_device(Vector4f *imageData_out, Vector2i newDims, const Vector4f *imageData_in, Vector2i oldDims)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x > newDims.x - 1 || y > newDims.y - 1) return;

	filterSubsampleWithHoles(imageData_out, x, y, newDims, imageData_in, oldDims);
}

__global__ void gradientX_device(Vector4s *grad, const Vector4u *image, Vector2i imgSize)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < 1 || x > imgSize.x - 2 || y < 1 || y > imgSize.y - 2) return;

	gradientX(grad, x, y, image, imgSize);
}

__global__ void gradientY_device(Vector4s *grad, const Vector4u *image, Vector2i imgSize)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < 1 || x > imgSize.x - 2 || y < 1 || y > imgSize.y - 2) return;

	gradientY(grad, x, y, image, imgSize);
}

__global__ void gradientXY_device(Vector2f *grad, const float *image, Vector2i imgSize)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < 1 || x > imgSize.x - 2 || y < 1 || y > imgSize.y - 2) return;

	gradientXY(grad, x, y, image, imgSize);
}

__global__ void countValidDepths_device(const float *imageData_in, int imgSizeTotal, int *counterTempData_device)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int locId_local = threadIdx.x;

	__shared__ int dim_shared[256];
	//__shared__ bool should_prefix;

	//should_prefix = false;
	//__syncthreads();

	bool isValidPoint = false;

	if (i < imgSizeTotal)
	{
		if (imageData_in[i] > 0.0f) isValidPoint = true;
	}

	//__syncthreads();
	//if (!should_prefix) return;

	dim_shared[locId_local] = isValidPoint;
	__syncthreads();

	if (locId_local < 128) dim_shared[locId_local] += dim_shared[locId_local + 128];
	__syncthreads();
	if (locId_local < 64) dim_shared[locId_local] += dim_shared[locId_local + 64];
	__syncthreads();

	if (locId_local < 32) warpReduce(dim_shared, locId_local);

	if (locId_local == 0) atomicAdd(counterTempData_device, dim_shared[locId_local]);
}

__global__ void kernelBilateral5x5( Vector2i imgSize, const float *data, float* filtered_data, float depthUncertaintyCoef)
{
    int width = imgSize.width;
    int height = imgSize.height;

    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    if( x >= width || y >= height ){
        return;
    }

//    float smoothDepth = 0;
//    float totalCoef = 0;

    float refDepth = data[y*width+x];

    //float depthSigma = depthUncertaintyCoef * refDepth * refDepth * 0.5f;
    float depthSigma = depthUncertaintyCoef * refDepth * refDepth * 1e-1;
    //float depthSigma = depthUncertaintyCoef * refDepth * refDepth * 2.f;
    float depthVariance = depthSigma * depthSigma;
    float depthPrecision = 1.f / (2 * depthVariance);

    filtered_data[y*width+x] = bilateral5x5_shared(x,y,width,data,depthPrecision);
}