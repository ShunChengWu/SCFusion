#include "hip/hip_runtime.h"
// Copyright 2014-2017 Oxford University Innovation Limited and the authors of InfiniTAM

#include "ITMExtendedTracker_CUDA.h"
#include "../../Utils/ITMCUDAUtils.h"
#include "../Shared/ITMExtendedTracker_Shared.h"
#include <ORUtils/CUDADefines.h>

using namespace ITMLib;

struct ITMExtendedTracker_CUDA::AccuCell {
	int numPoints;
	float f;
	float g[6];
	float h[6+5+4+3+2+1];
};

struct ITMExtendedTracker_KernelParameters_Depth {
	ITMExtendedTracker_CUDA::AccuCell *accu;
	float *depth;
	Matrix4f approxInvPose;
	Vector4f *pointsMap;
	Vector4f *normalsMap;
	Vector4f sceneIntrinsics;
	Vector2i sceneImageSize;
	Matrix4f scenePose;
	Vector4f viewIntrinsics;
	Vector2i viewImageSize;
	float spaceThresh;
	float viewFrustum_min, viewFrustum_max;
	float tukeyCutOff;
	int framesToSkip, framesToWeight;
};

struct ITMExtendedTracker_KernelParameters_RGB {
	ITMExtendedTracker_CUDA::AccuCell *accu;
	const Vector4f *points_curr;
	const Vector2f *gradients;
	const float *intensities_curr;
	const float *intensities_prev;
	Vector2i imageSize_rgb;
	Vector2i imageSize_depth;
	Matrix4f approxInvPose;
	Matrix4f scenePose;
	Vector4f projParams_depth;
	Vector4f projParams_rgb;
	float colourThresh;
	float minGradient;
	float viewFrustum_min, viewFrustum_max;
	float tukeyCutOff;
};

template<bool shortIteration, bool rotationOnly, bool useWeights>
__global__ void exDepthTrackerOneLevel_g_rt_device(ITMExtendedTracker_KernelParameters_Depth para);

template<bool shortIteration, bool rotationOnly>
__global__ void exRGBTrackerOneLevel_g_rt_device(ITMExtendedTracker_KernelParameters_RGB para);

__global__ void exRGBTrackerProjectPrevImage_device(Vector4f *out_points, float *out_rgb, const float *in_rgb, const float *in_points, Vector2i imageSize, Vector2i sceneSize, Vector4f intrinsics_depth, Vector4f intrinsics_rgb, Matrix4f scenePose);

// host methods

ITMExtendedTracker_CUDA::ITMExtendedTracker_CUDA(Vector2i imgSize_d,
												 Vector2i imgSize_rgb,
												 bool useDepth,
												 bool useColour,
												 float colourWeight,
												 TrackerIterationType *trackingRegime,
												 int noHierarchyLevels,
												 float terminationThreshold,
												 float failureDetectorThreshold,
												 float viewFrustum_min,
												 float viewFrustum_max,
												 float minColourGradient,
												 float tukeyCutOff,
												 int framesToSkip,
												 int framesToWeight,
												 const ITMLowLevelEngine *lowLevelEngine)
	: ITMExtendedTracker(imgSize_d,
						 imgSize_rgb,
						 useDepth,
						 useColour,
						 colourWeight,
						 trackingRegime,
						 noHierarchyLevels,
						 terminationThreshold,
						 failureDetectorThreshold,
						 viewFrustum_min,
						 viewFrustum_max,
						 minColourGradient,
						 tukeyCutOff,
						 framesToSkip,
						 framesToWeight,
						 lowLevelEngine,
						 MEMORYDEVICE_CUDA)
{
	ORcudaSafeCall(hipHostMalloc((void**)&accu_host, sizeof(AccuCell)));
	ORcudaSafeCall(hipMalloc((void**)&accu_device, sizeof(AccuCell)));
}

ITMExtendedTracker_CUDA::~ITMExtendedTracker_CUDA(void)
{
	ORcudaSafeCall(hipHostFree(accu_host));
	ORcudaSafeCall(hipFree(accu_device));
}

int ITMExtendedTracker_CUDA::ComputeGandH_Depth(float &f, float *nabla, float *hessian, Matrix4f approxInvPose)
{
	Vector2i sceneImageSize = sceneHierarchyLevel_Depth->pointsMap->noDims;
	Vector2i viewImageSize = viewHierarchyLevel_Depth->depth->noDims;

	if (currentIterationType == TRACKER_ITERATION_NONE) return 0;

	bool shortIteration = currentIterationType == TRACKER_ITERATION_ROTATION
						  || currentIterationType == TRACKER_ITERATION_TRANSLATION;

	int noPara = shortIteration ? 3 : 6;

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)viewImageSize.x / (float)blockSize.x), (int)ceil((float)viewImageSize.y / (float)blockSize.y));

	ORcudaSafeCall(hipMemset(accu_device, 0, sizeof(AccuCell)));

	ITMExtendedTracker_KernelParameters_Depth args;
	args.accu = accu_device;
	args.depth = viewHierarchyLevel_Depth->depth->GetData(MEMORYDEVICE_CUDA);
	args.approxInvPose = approxInvPose;
	args.pointsMap = sceneHierarchyLevel_Depth->pointsMap->GetData(MEMORYDEVICE_CUDA);
	args.normalsMap = sceneHierarchyLevel_Depth->normalsMap->GetData(MEMORYDEVICE_CUDA);
	args.sceneIntrinsics = sceneHierarchyLevel_Depth->intrinsics;
	args.sceneImageSize = sceneImageSize;
	args.scenePose = scenePose;
	args.viewIntrinsics = viewHierarchyLevel_Depth->intrinsics;
	args.viewImageSize = viewHierarchyLevel_Depth->depth->noDims;
	args.spaceThresh = spaceThresh[currentLevelId];
	args.viewFrustum_min = viewFrustum_min;
	args.viewFrustum_max = viewFrustum_max;
	args.tukeyCutOff = tukeyCutOff;
	args.framesToSkip = framesToSkip;
	args.framesToWeight = framesToWeight;

	if (framesProcessed < 100)
	{
		switch (currentIterationType)
		{
		case TRACKER_ITERATION_ROTATION:
			exDepthTrackerOneLevel_g_rt_device<true, true, false> << <gridSize, blockSize >> >(args);
			ORcudaKernelCheck;
			break;
		case TRACKER_ITERATION_TRANSLATION:
			exDepthTrackerOneLevel_g_rt_device<true, false, false> << <gridSize, blockSize >> >(args);
			ORcudaKernelCheck;
			break;
		case TRACKER_ITERATION_BOTH:
			exDepthTrackerOneLevel_g_rt_device<false, false, false> << <gridSize, blockSize >> >(args);
			ORcudaKernelCheck;
			break;
		default: break;
		}
	}
	else
	{
		switch (currentIterationType)
		{
		case TRACKER_ITERATION_ROTATION:
			exDepthTrackerOneLevel_g_rt_device<true, true, true> << <gridSize, blockSize >> >(args);
			ORcudaKernelCheck;
			break;
		case TRACKER_ITERATION_TRANSLATION:
			exDepthTrackerOneLevel_g_rt_device<true, false, true> << <gridSize, blockSize >> >(args);
			ORcudaKernelCheck;
			break;
		case TRACKER_ITERATION_BOTH:
			exDepthTrackerOneLevel_g_rt_device<false, false, true> << <gridSize, blockSize >> >(args);
			ORcudaKernelCheck;
			break;
		default: break;
		}
	}

	ORcudaSafeCall(hipMemcpy(accu_host, accu_device, sizeof(AccuCell), hipMemcpyDeviceToHost));

	// Copy the lower triangular part of the matrix.
	for (int r = 0, counter = 0; r < noPara; r++)
		for (int c = 0; c <= r; c++, counter++)
			hessian[r + c * 6] = accu_host->h[counter];

	// Transpose to fill the upper triangle.
	for (int r = 0; r < noPara; ++r)
		for (int c = r + 1; c < noPara; c++)
			hessian[r + c * 6] = hessian[c + r * 6];

	memcpy(nabla, accu_host->g, noPara * sizeof(float));

	f = accu_host->f;

	return accu_host->numPoints;
}

int ITMExtendedTracker_CUDA::ComputeGandH_RGB(float &f, float *nabla, float *hessian, Matrix4f approxInvPose)
{
	Vector2i imageSize_depth = viewHierarchyLevel_Depth->depth->noDims;
	Vector2i imageSize_rgb = viewHierarchyLevel_Intensity->intensity_prev->noDims;

	if (currentIterationType == TRACKER_ITERATION_NONE) return 0;

	bool shortIteration = currentIterationType == TRACKER_ITERATION_ROTATION
						  || currentIterationType == TRACKER_ITERATION_TRANSLATION;

	int noPara = shortIteration ? 3 : 6;

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)imageSize_depth.x / (float)blockSize.x), (int)ceil((float)imageSize_depth.y / (float)blockSize.y));

	ORcudaSafeCall(hipMemset(accu_device, 0, sizeof(AccuCell)));

	ITMExtendedTracker_KernelParameters_RGB args;
	args.accu = accu_device;
	args.points_curr = reprojectedPointsLevel->data->GetData(MEMORYDEVICE_CUDA);
	args.intensities_curr = projectedIntensityLevel->data->GetData(MEMORYDEVICE_CUDA);
	args.intensities_prev = viewHierarchyLevel_Intensity->intensity_prev->GetData(MEMORYDEVICE_CUDA);
	args.gradients = viewHierarchyLevel_Intensity->gradients->GetData(MEMORYDEVICE_CUDA);
	args.imageSize_rgb = imageSize_rgb;
	args.imageSize_depth = imageSize_depth;
	args.approxInvPose = approxInvPose;
	args.scenePose = depthToRGBTransform * scenePose;
	args.projParams_depth = viewHierarchyLevel_Depth->intrinsics;
	args.projParams_rgb = viewHierarchyLevel_Intensity->intrinsics;
	args.colourThresh = colourThresh[currentLevelId];
	args.minGradient = minColourGradient;
	args.viewFrustum_min = viewFrustum_min;
	args.viewFrustum_max = viewFrustum_max;
	args.tukeyCutOff = tukeyCutOff;

	switch (currentIterationType)
	{
	case TRACKER_ITERATION_ROTATION:
		exRGBTrackerOneLevel_g_rt_device<true, true> << <gridSize, blockSize >> >(args);
		ORcudaKernelCheck;
		break;
	case TRACKER_ITERATION_TRANSLATION:
		exRGBTrackerOneLevel_g_rt_device<true, false> << <gridSize, blockSize >> >(args);
		ORcudaKernelCheck;
		break;
	case TRACKER_ITERATION_BOTH:
		exRGBTrackerOneLevel_g_rt_device<false, false> << <gridSize, blockSize >> >(args);
		ORcudaKernelCheck;
		break;
	default: break;
	}

	ORcudaSafeCall(hipMemcpy(accu_host, accu_device, sizeof(AccuCell), hipMemcpyDeviceToHost));

	// Copy the lower triangular part of the matrix.
	for (int r = 0, counter = 0; r < noPara; r++)
		for (int c = 0; c <= r; c++, counter++)
			hessian[r + c * 6] = accu_host->h[counter];

	// Transpose to fill the upper triangle.
	for (int r = 0; r < noPara; ++r)
		for (int c = r + 1; c < noPara; c++)
			hessian[r + c * 6] = hessian[c + r * 6];

	memcpy(nabla, accu_host->g, noPara * sizeof(float));

	f = accu_host->f;

	return accu_host->numPoints;
}

void ITMExtendedTracker_CUDA::ProjectCurrentIntensityFrame(ITMFloat4Image *points_out,
														   ITMFloatImage *intensity_out,
														   const ITMFloatImage *intensity_in,
														   const ITMFloatImage *depth_in,
														   const Vector4f &intrinsics_depth,
														   const Vector4f &intrinsics_rgb,
														   const Matrix4f &scenePose)
{
	const Vector2i imageSize_rgb = intensity_in->noDims;
	const Vector2i imageSize_depth = depth_in->noDims; // Also the size of the projected image

	points_out->ChangeDims(imageSize_depth); // Actual reallocation should happen only once per run.
	intensity_out->ChangeDims(imageSize_depth); // Actual reallocation should happen only once per run.

	const float *depths = depth_in->GetData(MEMORYDEVICE_CUDA);
	const float *intensityIn = intensity_in->GetData(MEMORYDEVICE_CUDA);
	Vector4f *pointsOut = points_out->GetData(MEMORYDEVICE_CUDA);
	float *intensityOut = intensity_out->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)imageSize_depth.x / (float)blockSize.x), (int)ceil((float)imageSize_depth.y / (float)blockSize.y));

	exRGBTrackerProjectPrevImage_device<<<gridSize, blockSize>>>(pointsOut, intensityOut, intensityIn, depths, imageSize_rgb, imageSize_depth, intrinsics_rgb, intrinsics_depth, scenePose);
	ORcudaKernelCheck;
}

// device functions
template<bool shortIteration, bool rotationOnly, bool useWeights>
__device__ void exDepthTrackerOneLevel_g_rt_device_main(ITMExtendedTracker_CUDA::AccuCell *accu, float *depth,
	Matrix4f approxInvPose, Vector4f *pointsMap, Vector4f *normalsMap, Vector4f sceneIntrinsics, Vector2i sceneImageSize, Matrix4f scenePose,
	Vector4f viewIntrinsics, Vector2i viewImageSize, float spaceThresh, float viewFrustum_min, float viewFrustum_max,
	float tukeyCutOff, int framesToSkip, int framesToWeight)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	int locId_local = threadIdx.x + threadIdx.y * blockDim.x;

	__shared__ float dim_shared1[256];
	__shared__ float dim_shared2[256];
	__shared__ float dim_shared3[256];
	__shared__ bool should_prefix;

	should_prefix = false;
	__syncthreads();

	const int noPara = shortIteration ? 3 : 6;
	const int noParaSQ = shortIteration ? 3 + 2 + 1 : 6 + 5 + 4 + 3 + 2 + 1;
	float A[noPara]; float b; float depthWeight = 1.0f;

	bool isValidPoint = false;

	if (x < viewImageSize.x && y < viewImageSize.y)
	{
		isValidPoint = computePerPointGH_exDepth_Ab<shortIteration, rotationOnly, useWeights>(A, b, x, y, depth[x + y * viewImageSize.x], depthWeight,
			viewImageSize, viewIntrinsics, sceneImageSize, sceneIntrinsics, approxInvPose, scenePose, pointsMap, normalsMap, spaceThresh,
			viewFrustum_min, viewFrustum_max, tukeyCutOff, framesToSkip, framesToWeight);

		if (isValidPoint) should_prefix = true;
	}

	if (!isValidPoint) {
		for (int i = 0; i < noPara; i++) A[i] = 0.0f;
		b = 0.0f;
	}

	__syncthreads();

	if (!should_prefix) return;

	{ //reduction for noValidPoints
		dim_shared1[locId_local] = isValidPoint;
		__syncthreads();

		if (locId_local < 128) dim_shared1[locId_local] += dim_shared1[locId_local + 128];
		__syncthreads();
		if (locId_local < 64) dim_shared1[locId_local] += dim_shared1[locId_local + 64];
		__syncthreads();

		if (locId_local < 32) warpReduce(dim_shared1, locId_local);

		if (locId_local == 0) atomicAdd(&(accu->numPoints), (int)dim_shared1[locId_local]);
	}

	__syncthreads();

	{ //reduction for energy function value
		dim_shared1[locId_local] = rho(b, spaceThresh) * depthWeight;
		__syncthreads();

		if (locId_local < 128) dim_shared1[locId_local] += dim_shared1[locId_local + 128];
		__syncthreads();
		if (locId_local < 64) dim_shared1[locId_local] += dim_shared1[locId_local + 64];
		__syncthreads();

		if (locId_local < 32) warpReduce(dim_shared1, locId_local);

		if (locId_local == 0) atomicAdd(&(accu->f), dim_shared1[locId_local]);
	}

	__syncthreads();

	//reduction for nabla
	for (unsigned char paraId = 0; paraId < noPara; paraId+=3)
	{
		dim_shared1[locId_local] = rho_deriv(b, spaceThresh) * depthWeight * A[paraId + 0];
		dim_shared2[locId_local] = rho_deriv(b, spaceThresh) * depthWeight * A[paraId + 1];
		dim_shared3[locId_local] = rho_deriv(b, spaceThresh) * depthWeight * A[paraId + 2];
		__syncthreads();

		if (locId_local < 128) {
			dim_shared1[locId_local] += dim_shared1[locId_local + 128];
			dim_shared2[locId_local] += dim_shared2[locId_local + 128];
			dim_shared3[locId_local] += dim_shared3[locId_local + 128];
		}
		__syncthreads();
		if (locId_local < 64) {
			dim_shared1[locId_local] += dim_shared1[locId_local + 64];
			dim_shared2[locId_local] += dim_shared2[locId_local + 64];
			dim_shared3[locId_local] += dim_shared3[locId_local + 64];
		}
		__syncthreads();

		if (locId_local < 32) {
			warpReduce(dim_shared1, locId_local);
			warpReduce(dim_shared2, locId_local);
			warpReduce(dim_shared3, locId_local);
		}
		__syncthreads();

		if (locId_local == 0) {
			atomicAdd(&(accu->g[paraId+0]), dim_shared1[0]);
			atomicAdd(&(accu->g[paraId+1]), dim_shared2[0]);
			atomicAdd(&(accu->g[paraId+2]), dim_shared3[0]);
		}
	}

	__syncthreads();

	float localHessian[noParaSQ];
#if (defined(__HIPCC__) && defined(__CUDA_ARCH__)) || (defined(__METALC__))
#pragma unroll
#endif
	for (unsigned char r = 0, counter = 0; r < noPara; r++)
	{
#if (defined(__HIPCC__) && defined(__CUDA_ARCH__)) || (defined(__METALC__))
#pragma unroll
#endif
		for (int c = 0; c <= r; c++, counter++) localHessian[counter] = rho_deriv2(b, spaceThresh) * depthWeight * A[r] * A[c];
	}

	//reduction for hessian
	for (unsigned char paraId = 0; paraId < noParaSQ; paraId+=3)
	{
		dim_shared1[locId_local] = localHessian[paraId+0];
		dim_shared2[locId_local] = localHessian[paraId+1];
		dim_shared3[locId_local] = localHessian[paraId+2];
		__syncthreads();

		if (locId_local < 128) {
			dim_shared1[locId_local] += dim_shared1[locId_local + 128];
			dim_shared2[locId_local] += dim_shared2[locId_local + 128];
			dim_shared3[locId_local] += dim_shared3[locId_local + 128];
		}
		__syncthreads();
		if (locId_local < 64) {
			dim_shared1[locId_local] += dim_shared1[locId_local + 64];
			dim_shared2[locId_local] += dim_shared2[locId_local + 64];
			dim_shared3[locId_local] += dim_shared3[locId_local + 64];
		}
		__syncthreads();

		if (locId_local < 32) {
			warpReduce(dim_shared1, locId_local);
			warpReduce(dim_shared2, locId_local);
			warpReduce(dim_shared3, locId_local);
		}
		__syncthreads();

		if (locId_local == 0) {
			atomicAdd(&(accu->h[paraId+0]), dim_shared1[0]);
			atomicAdd(&(accu->h[paraId+1]), dim_shared2[0]);
			atomicAdd(&(accu->h[paraId+2]), dim_shared3[0]);
		}
	}
}

template<bool shortIteration, bool rotationOnly>
__device__ void exRGBTrackerOneLevel_g_rt_device_main(
		ITMExtendedTracker_CUDA::AccuCell *accu,
		const Vector4f *points_curr,
		const float *intensities_prev,
		const Vector2f *gradients,
		const float *intensities_curr,
		const Matrix4f &approxInvPose,
		const Matrix4f &scenePose,
		const Vector4f &projParams_depth,
		const Vector4f &projParams_rgb,
		const Vector2i &imageSize_rgb,
		const Vector2i &imageSize_depth,
		float colourThresh,
		float minGradient,
		float viewFrustum_min,
		float viewFrustum_max,
		float tukeyCutoff
		)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	int locId_local = threadIdx.x + threadIdx.y * blockDim.x;

	__shared__ float dim_shared1[256];
	__shared__ float dim_shared2[256];
	__shared__ float dim_shared3[256];
	__shared__ bool should_prefix;

	should_prefix = false;
	__syncthreads();

	const int noPara = shortIteration ? 3 : 6;
	const int noParaSQ = shortIteration ? 3 + 2 + 1 : 6 + 5 + 4 + 3 + 2 + 1;
	float localHessian[noParaSQ];
	float localNabla[noPara];
	float localF;

	bool isValidPoint = false;

	if (x < imageSize_depth.x && y < imageSize_depth.y)
	{
		isValidPoint = computePerPointGH_exRGB_inv_Ab<shortIteration, rotationOnly>(
				localF,
				localNabla,
				localHessian,
				x,
				y,
				points_curr,
				intensities_curr,
				intensities_prev,
				gradients,
				imageSize_depth,
				imageSize_rgb,
				projParams_depth,
				projParams_rgb,
				approxInvPose,
				scenePose,
				colourThresh,
				minGradient,
				viewFrustum_min,
				viewFrustum_max,
				tukeyCutoff
				);

		if (isValidPoint) should_prefix = true;
	}

	if (!isValidPoint)
	{
		for (int i = 0; i < noParaSQ; i++) localHessian[i] = 0.0f;
		for (int i = 0; i < noPara; i++) localNabla[i] = 0.0f;
		localF = 0.0f;
	}

	__syncthreads();

	if (!should_prefix) return;

	{ //reduction for noValidPoints
		dim_shared1[locId_local] = isValidPoint;
		__syncthreads();

		if (locId_local < 128) dim_shared1[locId_local] += dim_shared1[locId_local + 128];
		__syncthreads();

		if (locId_local < 64) dim_shared1[locId_local] += dim_shared1[locId_local + 64];
		__syncthreads();

		if (locId_local < 32) warpReduce(dim_shared1, locId_local);

		if (locId_local == 0) atomicAdd(&(accu->numPoints), (int)dim_shared1[locId_local]);
	}

	__syncthreads();

	{ //reduction for energy function value
		dim_shared1[locId_local] = localF;
		__syncthreads();

		if (locId_local < 128) dim_shared1[locId_local] += dim_shared1[locId_local + 128];
		__syncthreads();
		if (locId_local < 64) dim_shared1[locId_local] += dim_shared1[locId_local + 64];
		__syncthreads();

		if (locId_local < 32) warpReduce(dim_shared1, locId_local);
		__syncthreads();

		if (locId_local == 0) atomicAdd(&(accu->f), dim_shared1[locId_local]);
	}

	__syncthreads();

	//reduction for nabla
	for (unsigned char paraId = 0; paraId < noPara; paraId += 3)
	{
		dim_shared1[locId_local] = localNabla[paraId + 0];
		dim_shared2[locId_local] = localNabla[paraId + 1];
		dim_shared3[locId_local] = localNabla[paraId + 2];
		__syncthreads();

		if (locId_local < 128) {
			dim_shared1[locId_local] += dim_shared1[locId_local + 128];
			dim_shared2[locId_local] += dim_shared2[locId_local + 128];
			dim_shared3[locId_local] += dim_shared3[locId_local + 128];
		}
		__syncthreads();
		if (locId_local < 64) {
			dim_shared1[locId_local] += dim_shared1[locId_local + 64];
			dim_shared2[locId_local] += dim_shared2[locId_local + 64];
			dim_shared3[locId_local] += dim_shared3[locId_local + 64];
		}
		__syncthreads();

		if (locId_local < 32) {
			warpReduce(dim_shared1, locId_local);
			warpReduce(dim_shared2, locId_local);
			warpReduce(dim_shared3, locId_local);
		}
		__syncthreads();

		if (locId_local == 0) {
			atomicAdd(&(accu->g[paraId + 0]), dim_shared1[0]);
			atomicAdd(&(accu->g[paraId + 1]), dim_shared2[0]);
			atomicAdd(&(accu->g[paraId + 2]), dim_shared3[0]);
		}
	}

	__syncthreads();

	//reduction for hessian
	for (unsigned char paraId = 0; paraId < noParaSQ; paraId += 3)
	{
		dim_shared1[locId_local] = localHessian[paraId + 0];
		dim_shared2[locId_local] = localHessian[paraId + 1];
		dim_shared3[locId_local] = localHessian[paraId + 2];
		__syncthreads();

		if (locId_local < 128) {
			dim_shared1[locId_local] += dim_shared1[locId_local + 128];
			dim_shared2[locId_local] += dim_shared2[locId_local + 128];
			dim_shared3[locId_local] += dim_shared3[locId_local + 128];
		}
		__syncthreads();
		if (locId_local < 64) {
			dim_shared1[locId_local] += dim_shared1[locId_local + 64];
			dim_shared2[locId_local] += dim_shared2[locId_local + 64];
			dim_shared3[locId_local] += dim_shared3[locId_local + 64];
		}
		__syncthreads();

		if (locId_local < 32) {
			warpReduce(dim_shared1, locId_local);
			warpReduce(dim_shared2, locId_local);
			warpReduce(dim_shared3, locId_local);
		}
		__syncthreads();

		if (locId_local == 0) {
			atomicAdd(&(accu->h[paraId + 0]), dim_shared1[0]);
			atomicAdd(&(accu->h[paraId + 1]), dim_shared2[0]);
			atomicAdd(&(accu->h[paraId + 2]), dim_shared3[0]);
		}
	}
}

template<bool shortIteration, bool rotationOnly, bool useWeights>
__global__ void exDepthTrackerOneLevel_g_rt_device(ITMExtendedTracker_KernelParameters_Depth para)
{
	exDepthTrackerOneLevel_g_rt_device_main<shortIteration, rotationOnly, useWeights>(para.accu, para.depth,
		para.approxInvPose, para.pointsMap, para.normalsMap, para.sceneIntrinsics, para.sceneImageSize, para.scenePose,
		para.viewIntrinsics, para.viewImageSize, para.spaceThresh, para.viewFrustum_min, para.viewFrustum_max,
		para.tukeyCutOff, para.framesToSkip, para.framesToWeight);
}

template<bool shortIteration, bool rotationOnly>
__global__ void exRGBTrackerOneLevel_g_rt_device(ITMExtendedTracker_KernelParameters_RGB para)
{
	exRGBTrackerOneLevel_g_rt_device_main<shortIteration, rotationOnly>(
			para.accu,
			para.points_curr,
			para.intensities_prev,
			para.gradients,
			para.intensities_curr,
			para.approxInvPose,
			para.scenePose,
			para.projParams_depth,
			para.projParams_rgb,
			para.imageSize_rgb,
			para.imageSize_depth,
			para.colourThresh,
			para.minGradient,
			para.viewFrustum_min,
			para.viewFrustum_max,
			para.tukeyCutOff
			);
}

__global__ void exRGBTrackerProjectPrevImage_device(Vector4f *out_points, float *out_rgb, const float *in_rgb, const float *in_points, Vector2i imageSize, Vector2i sceneSize, Vector4f intrinsics_depth, Vector4f intrinsics_rgb, Matrix4f scenePose)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	projectPoint_exRGB(x, y, out_points, out_rgb, in_rgb, in_points, imageSize, sceneSize, intrinsics_depth, intrinsics_rgb, scenePose);
}
